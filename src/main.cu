#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
// For random number generator
#include <time.h>
// For square root
#include <math.h>

#include "helperfuncs.h"
#include "picture.h"
#include "floatvolume.h"

void setDiffVolumeSerial(struct FloatVolume *fv, struct Picture *picture1,
  struct Picture *picture2) {
  unsigned i, j, k;
  float *p1c, *p2c;

  // If pictures have differing dimensions, then quit
  if(picture1->width != picture2->width ||
    picture1->height != picture2->height) {
    printf("Pictures have different dimensions. Exiting setDiffVolmeSerial\n");
    return;
  }

  fv->height = picture1->height;
  fv->width = picture1->width;
  fv->depth = picture1->width;

  fv->contents = (float *) malloc(sizeof(float) * fv->height * fv->width *
    fv->depth);

  for(i = 0; i < fv->height; i++) {
    for(j = 0; j < fv->width; j++) {
      for(k = 0; k < fv->depth; k++) {
        // Get the index of the pixel of the first picture
        p1c = picture1->colors + toIndex2D(i, j, picture1->width) * 4;
        // Get the index of the pixel of the second picture
        p2c = picture2->colors + toIndex2D(i, k, picture2->width) * 4;

        // Insert the distance between these two colors into the float volume
        *(fv->contents + toIndex3D(i, j, fv->width, k, fv->depth)) =
          diffColor(p1c, p2c);

        /* *(fv->contents + toIndex3D(i, j, fv->width, k, fv->depth)) =
          toIndex3D(i, j, fv->width, k, fv->depth); */
      }
    }
  }
}

__global__ void setDiffVolumeKernel(float *d_fv, float *d_picture1,
  float *d_picture2, unsigned picWidth, unsigned picHeight) {
  __shared__ float p1_section[10 * 10 * 4];
  __shared__ float p2_section[10 * 10 * 4];
  unsigned i;

  // This thread's position in its block's subsection of the float volume
  unsigned sx, sy, sz;
  // Dimensions of the grid
  unsigned gx, gy, gz;
  // Position of this thread's block
  unsigned bx, by, bz;
  // This thread's position in the entire float volume
  unsigned vx, vy, vz;
  // The location of the colors that this thread will be comparing
  unsigned c1, c2;

  // Get the position of this thread in its subsection
  sz = threadIdx.x % 10;
  sy = threadIdx.x / 100;
  sx = (threadIdx.x % 100) / 10;

  // Get the dimensions of the grid
  gz = picWidth / 10;
  if(picWidth % 10) gz++;
  gy = picHeight / 10;
  if(picHeight % 10) gy++;
  gx = picWidth / 10;
  if(picWidth % 10) gx++;

  // Get the position of this thread's block
  bz = blockIdx.x % gz;
  by = blockIdx.x / (gx * gz);
  bx = (blockIdx.x % (gx * gz)) / gz;

  // Get the position of this thread in entire float volume
  vx = sx + 10 * bx;
  vy = sy + 10 * by;
  vz = sz + 10 * bz;

  // Copy subpicture to shared memory

  // See if this thread needs to copy from picture 1
  // picture 1 covers width * height

  // If the float volume z of this thread is zero,
  // then it needs to copy from picture 1
  if(sz == 0) {
    // Check if this thread will get a pixel not in the picture
    if(vx < picWidth && vy < picHeight) {
      for(i = 0; i < 4; i++) {
        p1_section[(sx + sy * 10) * 4 + i] =
          d_picture1[(vx + vy * picWidth) * 4 + i];
      }
    }
  }

  // See if this thread needs to copy from picture 2
  // picture 2 covers depth * height

  // If the float volume x of this thread is zero,
  // then it needs to copy from picture 2
  if(sx == 0) {
    // Check if this thread will get a pixel not in the picture
    if(vz < picWidth && vy < picHeight) {
      for(i = 0; i < 4; i++) {
        p2_section[(sz + sy * 10) * 4 + i] =
          d_picture2[(vz + vy * picWidth) * 4 + i];
      }
    }
  }

  __syncthreads();
  // Now each of d_picture1 and d_picture2 are properly filled out

  // Write difference into float volume
  if(vx < picWidth && vy < picHeight && vz < picWidth) {
    c1 = (sx + sy * 10) * 4;
    c2 = (sz + sy * 10) * 4;
    d_fv[vz + vx * picWidth + vy * picWidth * picWidth] =
      sqrtf(
        powf(p1_section[c1 + 0] - p2_section[c2 + 0], 2.f) +
        powf(p1_section[c1 + 1] - p2_section[c2 + 1], 2.f) +
        powf(p1_section[c1 + 2] - p2_section[c2 + 2], 2.f) +
        powf(p1_section[c1 + 3] - p2_section[c2 + 3], 2.f)
      );
  }
}

// TODO : make as parameters - max amount of memory, max number of blocks
void setDiffVolumeParallel(struct FloatVolume *fv, struct Picture *picture1,
  struct Picture *picture2) {
  // Memory locations of float arrays on the GPU
  float *d_fv, *d_picture1, *d_picture2;
  int fvDataLen;
  unsigned num_blocks, gdim[3];

  // If pictures have differing dimensions, then quit
  if(picture1->width != picture2->width ||
    picture2->height != picture2->height) {
    printf(
      "Pictures have different dimensions. Exiting setDiffVolumeParallel\n");
    return;
  }

  // Create the FloatVolume
  fv->height = picture1->height;
  fv->width = picture1->width;
  fv->depth = picture1->width;

  fvDataLen = fv->height * fv->width * fv->depth;

  fv->contents = (float *) malloc(sizeof(float) * fvDataLen);

  // Allocate space on the GPU
  hipMalloc((void **) &d_fv, fvDataLen * sizeof(float));
  hipMalloc((void **) &d_picture1, picture1->width * picture1->height *
    sizeof(float) * 4);
  hipMalloc((void **) &d_picture2, picture1->width * picture1->height *
    sizeof(float) * 4);

  // Give the pictures to the GPU
  // Params: destination, source, size of data to be copied, operation
  hipMemcpy(d_picture1, picture1->colors,
    picture1->width * picture1->height * 4 * sizeof(float),
    hipMemcpyHostToDevice);
  hipMemcpy(d_picture2, picture2->colors,
    picture1->width * picture1->height * 4 * sizeof(float),
    hipMemcpyHostToDevice);

  // Kernel stuff
  // 1000 threads per block
  // So get 10 x 10 subset of each picture, with 4 colors each

  // Get the number of blocks this program will use
  // TODO : Assume that the maximum number of blocks that can run
  //   at the same time is unlimited

  // y, then x, then z
  gdim[0] = fv->height / 10;
  if(fv->height % 10)
    gdim[0] = gdim[0] + 1;
  gdim[1] = fv->width / 10;
  if(fv->width % 10)
    gdim[1] = gdim[1] + 1;
  gdim[2] = fv->depth / 10;
  if(fv->depth % 10)
    gdim[2] = gdim[2] + 1;
  num_blocks = gdim[0] * gdim[1] * gdim[2];
  dim3 dimGrid(num_blocks);
  dim3 dimBlock(1000);

  // Do it
  setDiffVolumeKernel<<<dimGrid, dimBlock>>>(d_fv, d_picture1, d_picture2,
    picture1->width, picture1->height);

  // Copy the float volume back into host memory
  hipMemcpy(fv->contents, d_fv, fvDataLen * sizeof(float),
    hipMemcpyDeviceToHost);

  // Clear memory
  hipFree(d_fv);
  hipFree(d_picture1);
  hipFree(d_picture2);
}

// Inefficient but working
void setBigDiffVolumeParallel(struct FloatVolume *fv, struct Picture *picture1,
  struct Picture *picture2, unsigned subpicture_height) {
  unsigned numIterations, i;
  struct FloatVolume subfloatvolume;
  struct Picture subpicture1, subpicture2;
  unsigned subpicture_size, subfloatvolume_size;
  unsigned lastIterHeight, last_subpicture_size, last_subfloatvolume_size;
  int fvDataLen;

  // If pictures have differing dimensions, then quit
  if(picture1->width != picture2->width ||
    picture1->height != picture2->height) {
    printf(
      "Pictures have different dimensions. Exiting setBigDiffVolumeParallel");
    return;
  }

  // Check if bad subpicture_height value
  if(subpicture_height > picture1->height) {
    printf("subpicture_height > picture1->height\n");
    return;
  }

  // Allocate space for the final float volume
  fv->height = picture1->height;
  fv->width = picture1->width;
  fv->depth = picture1->width;
  fvDataLen = fv->height * fv->width * fv->depth;
  fv->contents = (float *) malloc(sizeof(float) * fvDataLen);

  // Clear float volume for testing
  for(i = 0; i < fvDataLen; i++) {
    *(fv->contents + i) = 0.f;
  }

  // Allocate space for each of the subpictures
  // (subfloatvolume will be allocated in setDiffVolumeParallel)
  subpicture1.width = picture1->width;
  subpicture1.height = subpicture_height;
  subpicture1.colors = (float *) malloc(sizeof(float) * subpicture1.width *
    subpicture1.height * 4); // RGBA
  // Note: dimensions of picture1 and picture2 are the same
  subpicture2.width = picture2->width;
  subpicture2.height = subpicture_height;
  subpicture2.colors = (float *) malloc(sizeof(float) * subpicture2.width *
    subpicture2.height * 4); // RGBA

  numIterations = picture1->height / subpicture_height +
    ((picture1->height % subpicture_height) > 0);
  // How many 32 bit floats are in one subpicture
  subpicture_size = subpicture1.height * subpicture1.width * 4; // RGBA
  // How many 32 bit floats are in one subfloatvolume
  subfloatvolume_size = subpicture1.height * subpicture1.width *
    subpicture2.width;

  /* printf("--- Info ---\n");
  printf("subpicture1.dim: %u, %u\n", subpicture1.width, subpicture1.height);
  printf("subpicture2.dim: %u, %u\n", subpicture2.width, subpicture2.height);
  printf("numIterations: %u\n", numIterations);
  printf("subpicture_size: %u\n", subpicture_size);
  printf("subfloatvolume_size: %u\n", subfloatvolume_size); */

  printf("numIterations: %u\n", numIterations);
  // numIterations - 1 because last iteration is a special case
  for(i = 0; i < numIterations - 1; i++) {
    // Load the subpictures
    memcpy(subpicture1.colors, picture1->colors + subpicture_size * i,
      subpicture_size * sizeof(float));
    memcpy(subpicture2.colors, picture2->colors + subpicture_size * i,
      subpicture_size * sizeof(float));

    // Call the normal diff volume function
    setDiffVolumeParallel(&subfloatvolume, &subpicture1, &subpicture2);

    // Copy the results of the subvolume into the final float volume
    // Float volume is allocated inside of this function
    memcpy(fv->contents + subfloatvolume_size * i, subfloatvolume.contents,
      subfloatvolume_size * sizeof(float));

    // Deallocate the subvolume
    free(subfloatvolume.contents);
  }
  printf("Entering last iteration...\n");

  free(subpicture1.colors);
  free(subpicture2.colors);

  // Take care of case where last iteration must process subpictures with
  //   smaller heights

  // Find out heights of subpictures
  if(picture1->height % subpicture_height) {
    lastIterHeight = picture1->height % subpicture_height;
    // subpicture1.height = picture1->height % subpicture_height;
    subpicture1.height = lastIterHeight;
    subpicture2.height = lastIterHeight;
  }

  // Reallocate subpictures
  subpicture1.colors = (float *) malloc(sizeof(float) * subpicture1.width *
    subpicture1.height * 4); // RGBA
  subpicture2.colors = (float *) malloc(sizeof(float) * subpicture2.width *
    subpicture2.height * 4); // RGBA

  // Recalculate sizes
  last_subpicture_size = subpicture1.height * subpicture1.width * 4; // RGBA
  last_subfloatvolume_size = subpicture1.height * subpicture1.width *
    subpicture2.width;

  // Load the subpictures
  memcpy(subpicture1.colors, picture1->colors + subpicture_size * i,
    last_subpicture_size * sizeof(float));
  memcpy(subpicture2.colors, picture2->colors + subpicture_size * i,
    last_subpicture_size * sizeof(float));

  // Call the normal diff volume Function
  setDiffVolumeParallel(&subfloatvolume, &subpicture1, &subpicture2);

  // Copy the results of the subvolume into the final float volume
  memcpy(fv->contents + subfloatvolume_size * i, subfloatvolume.contents,
    last_subfloatvolume_size * sizeof(float));
}

// Fill cells of path volume where x = 0 and y = 0
void setX0Y0(struct FloatVolume *pv, struct FloatVolume *dv) {
  unsigned i;
  for(i = 1; i < pv->depth; i++) {
    *(pv->contents + toIndex3D(0, 0, pv->width, i, pv->depth)) =
      *(dv->contents + toIndex3D(0, 0, pv->width, i, pv->depth)) +
      *(pv->contents + toIndex3D(0, 0, pv->width, i - 1, pv->depth));
  }
}

// Fill cells where z = 0 and y = 0
void setZ0Y0(struct FloatVolume *pv, struct FloatVolume *dv) {
  unsigned i;
  for(i = 1; i < pv->width; i++) {
    *(pv->contents + toIndex3D(0, i, pv->width, 0, pv->depth)) =
      *(dv->contents + toIndex3D(0, i, pv->width, 0, pv->depth)) +
      *(pv->contents + toIndex3D(0, i - 1, pv->width, 0, pv->depth));
  }
}

// Fill cells where z = 0 and x = 0
void setZ0X0(struct FloatVolume *pv, struct FloatVolume *dv) {
  unsigned i;
  for(i = 1; i < pv->height; i++) {
    *(pv->contents + toIndex3D(i, 0, pv->width, 0, pv->depth)) =
      *(dv->contents + toIndex3D(i, 0, pv->width, 0, pv->depth)) +
      *(pv->contents + toIndex3D(i - 1, 0, pv->width, 0, pv->depth));
  }
}

// Fill cells where x = 0, assuming Z0X0 and Z0Y0 are filled
void setX0(struct FloatVolume *pv, struct FloatVolume *dv) {
  float candidates2D[3], minCandidate;
  unsigned i, j;
  for(i = 1; i < pv->height; i++) {
    for(j = 1; j < pv->depth; j++) {
      candidates2D[0] =
        *(pv->contents + toIndex3D(i, 0, pv->width, j - 1, pv->depth));
      candidates2D[1] =
        *(pv->contents + toIndex3D(i - 1, 0, pv->width, j - 1, pv->depth));
      candidates2D[2] =
        *(pv->contents + toIndex3D(i - 1, 0, pv->width, j, pv->depth));

      minCandidate = candidates2D[0];
      if(candidates2D[1] < minCandidate)
        minCandidate = candidates2D[1];
      if(candidates2D[2] < minCandidate)
        minCandidate = candidates2D[2];

      *(pv->contents + toIndex3D(i, 0, pv->width, j, pv->depth)) =
        *(dv->contents + toIndex3D(i, 0, pv->width, j, pv->depth)) +
        minCandidate;
    }
  }
}

// Fill cells where y = 0, assuming Z0Y0 and X0Y0 are filled
void setY0(struct FloatVolume *pv, struct FloatVolume *dv) {
  float candidates2D[3], minCandidate;
  unsigned i, j;

  for(i = 1; i < pv->width; i++) {
    for(j = 1; j < pv->depth; j++) {
      candidates2D[0] =
        *(pv->contents + toIndex3D(0, i, pv->width, j - 1, pv->depth));
      candidates2D[1] =
        *(pv->contents + toIndex3D(0, i - 1, pv->width, j - 1, pv->depth));
      candidates2D[2] =
        *(pv->contents + toIndex3D(0, i - 1, pv->width, j, pv->depth));

      minCandidate = candidates2D[0];
      if(candidates2D[1] < minCandidate)
        minCandidate = candidates2D[1];
      if(candidates2D[2] < minCandidate)
        minCandidate = candidates2D[2];

      *(pv->contents + toIndex3D(0, i, pv->width, j, pv->depth)) =
        *(dv->contents + toIndex3D(0, i, pv->width, j, pv->depth)) +
        minCandidate;
    }
  }
}

// Fill cells where z = 0, assuming Z0Y0 and Z0X0 are filled
void setZ0(struct FloatVolume *pv, struct FloatVolume *dv) {
  float candidates2D[3], minCandidate;
  unsigned i, j;

  for(i = 1; i < pv->height; i++) {
    for(j = 1; j < pv->width; j++) {
      candidates2D[0] =
        *(pv->contents + toIndex3D(i, j - 1, pv->width, 0, pv->depth));
      candidates2D[1] =
        *(pv->contents + toIndex3D(i - 1, j - 1, pv->width, 0, pv->depth));
      candidates2D[2] =
        *(pv->contents + toIndex3D(i - 1, j, pv->width, 0, pv->depth));

      minCandidate = candidates2D[0];
      if(candidates2D[1] < minCandidate)
        minCandidate = candidates2D[1];
      if(candidates2D[2] < minCandidate)
        minCandidate = candidates2D[2];

      *(pv->contents + toIndex3D(i, j, pv->width, 0, pv->depth)) =
        *(dv->contents + toIndex3D(i, j, pv->width, 0, pv->depth)) +
        minCandidate;
    }
  }
}

// Function too thicc
void pathVolumeInit(struct FloatVolume *pv, struct FloatVolume *dv) {
  int fvDataLen;
  unsigned i, j;
  float candidates2D[3], minCandidate;

  // Create the FloatVolume
  pv->height = dv->height;
  pv->width = dv->width;
  pv->depth = dv->depth;

  fvDataLen = pv->height * pv->width * pv->depth;

  pv->contents = (float *) malloc(sizeof(float) * fvDataLen);

  // TESTING : Set all cells in fv2 to 0
  for(i = 0; i < pv->depth * pv->width * pv->height; i++) {
    *(pv->contents + i) = 0.f;
  }

  // Set the first cell
  *(pv->contents + 0) = *(dv->contents + 0);

  // Fill cells where x = 0 and y = 0
  setX0Y0(pv, dv);
  // Fill cells where z = 0 and y = 0
  setZ0Y0(pv, dv);
  // Fill cells where z = 0 and x = 0
  setZ0X0(pv, dv);
  // Fill cells where x = 0
  setX0(pv, dv);
  // Fill cells where y = 0
  setY0(pv, dv);
  // Fill cells where z = 0
  setZ0(pv, dv);
}

void setPathVolumeSerial(struct FloatVolume *pv, struct FloatVolume *dv) {
  unsigned i, j, k, l;
  float candidates3D[7], minCandidate;

  pathVolumeInit(pv, dv);

  // Finally fill in the remaining ones
  for(i = 1; i < pv->height; i++) {
    for(j = 1; j < pv->width; j++) {
      for(k = 1; k < pv->depth; k++) {
        candidates3D[0] = *(pv->contents +
          toIndex3D(i, j, pv->width, k - 1, pv->depth));
        candidates3D[1] = *(pv->contents +
          toIndex3D(i, j - 1, pv->width, k, pv->depth));
        candidates3D[2] = *(pv->contents +
          toIndex3D(i, j - 1, pv->width, k - 1, pv->depth));
        candidates3D[3] = *(pv->contents +
          toIndex3D(i - 1, j, pv->width, k, pv->depth));
        candidates3D[4] = *(pv->contents +
          toIndex3D(i - 1, j, pv->width, k - 1, pv->depth));
        candidates3D[5] = *(pv->contents +
          toIndex3D(i - 1, j - 1, pv->width, k, pv->depth));
        candidates3D[6] = *(pv->contents +
          toIndex3D(i - 1, j - 1, pv->width, k - 1, pv->depth));

        minCandidate = candidates3D[0];
        for(l = 1; l < 7; l++) {
          if(candidates3D[l] < minCandidate)
            minCandidate = candidates3D[l];
        }

        *(pv->contents + toIndex3D(i, j, pv->width, k, pv->depth)) =
          *(dv->contents + toIndex3D(i, j, pv->width, k, pv->depth)) +
          minCandidate;

        //*(pv->contents + toIndex3D(i, j, pv->width, k, pv->depth)) = 11.f;
      }
    }
  }
}

// Height, width, and height refer to the dimensions of the float volume
__global__ void setPathVolumeKernel(float *d_pv, float *d_dv, unsigned height,
  unsigned width, unsigned depth) {
  // The subvolume
  __shared__ float sv[11 * 11 * 11];
  float candidates3D[7], minCandidate;
  unsigned i, j;

  // This thread's position in its block's subsection of the float volume
  unsigned sx, sy, sz;
  // Dimensions of the grid
  unsigned gx, gy, gz;
  // Position of this thread's block
  unsigned bx, by, bz;
  // This thread's position in the entire float volume
  unsigned vx, vy, vz;

  // Get the position of this thread in its subsection
  sz = threadIdx.x % 10;
  sy = threadIdx.x / 100;
  sx = (threadIdx.x % 100) / 10;

  // Get the dimensions of the grid
  /* gz = (width - 1) / 10 + ((width - 1) % 10);
  gy = (height - 1) / 10 + ((height - 1) % 10);
  gx = (width - 1) / 10 + ((width - 1) % 10); */
  gz = (width - 1) / 10;
  if((width - 1) % 10) gz++;
  gy = (height - 1) / 10;
  if((height - 1) % 10) gy++;
  gx = (width - 1) / 10;
  if((width - 1) % 10) gx++;

  // Get the position of this thread's block
  bz = blockIdx.x % gz;
  by = blockIdx.x / (gx * gz);
  bx = (blockIdx.x % (gx * gz)) / gz;

  // Get the position of this thread in entire float volume
  vx = sx + 10 * bx + 1;
  vy = sy + 10 * by + 1;
  vz = sz + 10 * bz + 1;

  // ez brute force... for demo purposes

  // Make each thread do work over and over until subvolume is filled
  for(i = 0; i < 10 + (10 - 1) + (10 - 1); i++) {
    if(vy < height && vx < width && vz < depth) {
      candidates3D[0] = d_pv[vy * width * depth + vx * depth + (vz - 1)];
      candidates3D[1] = d_pv[vy * width * depth + (vx - 1) * depth + vz];
      candidates3D[2] = d_pv[vy * width * depth + (vx - 1) * depth + (vz - 1)];
      candidates3D[3] = d_pv[(vy - 1) * width * depth + vx * depth + vz];
      candidates3D[4] = d_pv[(vy - 1) * width * depth + vx * depth + (vz - 1)];
      candidates3D[5] = d_pv[(vy - 1) * width * depth + (vx - 1) * depth + vz];
      candidates3D[6] =
        d_pv[(vy - 1) * width * depth + (vx - 1) * depth + (vz - 1)];

      minCandidate = candidates3D[0];
      for(j = 1; j < 7; j++) {
        if(candidates3D[j] < minCandidate)
          minCandidate = candidates3D[j];
      }

      d_pv[vy * width * depth + vx * depth + vz] = minCandidate +
        d_dv[vy * width * depth + vx * depth + vz];
    }
    __syncthreads();
  }

  /* if(vy < height && vx < width && vz < depth) {
    d_pv[vy * width * depth + vx * depth + vz] = 11.f;
  } */
}

// TODO : Currently the easy implementation
void setPathVolumeParallel(struct FloatVolume *pv, struct FloatVolume *dv) {
  // Memory locations of float volumes on the GPU
  float *d_pv, *d_dv;
  int fvDataLen, i;
  // y, x, z
  unsigned gdim[3];
  unsigned num_blocks, num_iter;

  // Serial implementation
  pathVolumeInit(pv, dv);
  fvDataLen = pv->height * pv->width * pv->depth;

  // Allocate space on the GPU
  hipMalloc((void **) &d_pv, fvDataLen * sizeof(float));
  hipMalloc((void **) &d_dv, fvDataLen * sizeof(float));

  // Give the diff volume to the GPU
  hipMemcpy(d_dv, dv->contents, fvDataLen * sizeof(float),
    hipMemcpyHostToDevice);
  // Give incomplete path volume to the GPU
  hipMemcpy(d_pv, pv->contents, fvDataLen * sizeof(float),
    hipMemcpyHostToDevice);

  // Kernel stuff
  // 1000 threads per block
  // Each block will get 11 x 11 x 11 subset of dv

  // Get the number of blocks this program will use
  // TODO : Assume that the maximum number of lbocks that can run
  //   at the same time is unlimited
  // TODO : Breaks when one of the pictures' dimensions is not more than or
  //   equal to 2
  /* gdim[0] = ((pv->height - 1) / 10 + ((pv->height - 1) % 10));
  gdim[1] = ((pv->width - 1) / 10 + ((pv->width - 1) % 10));
  gdim[2] = ((pv->depth - 1) / 10 + ((pv->depth - 1) % 10)); */
  gdim[0] = (pv->height - 1) / 10;
  if((pv->height - 1) % 10)
    gdim[0] = gdim[0] + 1;
  gdim[1] = (pv->width - 1) / 10;
  if((pv->width - 1) % 10)
    gdim[1] = gdim[1] + 1;
  gdim[2] = (pv->depth - 1) / 10;
  if((pv->depth - 1) % 10)
    gdim[2] = gdim[2] + 1;

  num_blocks = gdim[0] * gdim[1] * gdim[2];
  // Houdini stuff (Manhattan distance + 1)
  num_iter = gdim[0] + (gdim[1] - 1) + (gdim[2] - 1);

  dim3 dimGrid(num_blocks);
  dim3 dimBlock(1000);

  /* printf("fvDataLen: %u\n", fvDataLen);
  printf("gdim: [%u, %u, %u]\n", gdim[0], gdim[1], gdim[2]);
  printf("num_blocks: %u\n", num_blocks);
  printf("num_iter: %u\n", num_iter); */

  for(i = 0; i < num_iter; i++) {
    // Each block will work on its own 10 x 10 x 10 portion
    // Will need info from the previous, so will need 11 x 11 x 11 portion

    // Dewit
    setPathVolumeKernel<<<dimGrid, dimBlock>>>(d_pv, d_dv, pv->height,
      pv->width, pv->depth);
  }

  // Copy the path volume back into host memory
  hipMemcpy(pv->contents, d_pv, fvDataLen * sizeof(float),
    hipMemcpyDeviceToHost);

  // Clear memory
  hipFree(d_dv);
  hipFree(d_pv);
}

// Given a float volume with a complete y = 0, construct rest of float volume
void setSmallPathVolumeParallel(struct FloatVolume *pv,
  struct FloatVolume *dv) {
  // Memory locations of float volumes on the GPU
  float *d_pv, *d_dv;
  int fvDataLen, i;
  // y, x, z
  unsigned gdim[3];
  unsigned num_blocks, num_iter;

  // Fill cells where x = 0 and z = 0
  setZ0X0(pv, dv);
  // Fill cells where x = 0
  setX0(pv, dv);
  // Fill cells where z = 0
  setZ0(pv, dv);

  fvDataLen = pv->height * pv->width * pv->depth;

  // Allocate space on the GPU
  hipMalloc((void **) &d_pv, fvDataLen * sizeof(float));
  hipMalloc((void **) &d_dv, fvDataLen * sizeof(float));

  // Give the diff volume to the GPU
  hipMemcpy(d_dv, dv->contents, fvDataLen * sizeof(float),
    hipMemcpyHostToDevice);
  // Give incomplete path volume to the GPU
  hipMemcpy(d_pv, pv->contents, fvDataLen * sizeof(float),
    hipMemcpyHostToDevice);

  // Kernel stuff
  // 1000 threads per block
  // Each block will get 11 x 11 x 11 subset of dv

  // Get the number of blocks this program will use
  // TODO : Assume that the maximum number of lbocks that can run
  //   at the same time is unlimited
  // TODO : Breaks when one of the pictures' dimensions is not more than or
  //   equal to 2
  /* gdim[0] = ((pv->height - 1) / 10 + ((pv->height - 1) % 10));
  gdim[1] = ((pv->width - 1) / 10 + ((pv->width - 1) % 10));
  gdim[2] = ((pv->depth - 1) / 10 + ((pv->depth - 1) % 10)); */
  gdim[0] = (pv->height - 1) / 10;
  if((pv->height - 1) % 10)
    gdim[0] = gdim[0] + 1;
  gdim[1] = (pv->width - 1) / 10;
  if((pv->width - 1) % 10)
    gdim[1] = gdim[1] + 1;
  gdim[2] = (pv->depth - 1) / 10;
  if((pv->depth - 1) % 10)
    gdim[2] = gdim[2] + 1;

  num_blocks = gdim[0] * gdim[1] * gdim[2];
  // Houdini stuff (Manhattan distance + 1)
  num_iter = gdim[0] + (gdim[1] - 1) + (gdim[2] - 1);

  dim3 dimGrid(num_blocks);
  dim3 dimBlock(1000);

  /* printf("fvDataLen: %u\n", fvDataLen);
  printf("gdim: [%u, %u, %u]\n", gdim[0], gdim[1], gdim[2]);
  printf("num_blocks: %u\n", num_blocks);
  printf("num_iter: %u\n", num_iter); */

  for(i = 0; i < num_iter; i++) {
    // Each block will work on its own 10 x 10 x 10 portion
    // Will need info from the previous, so will need 11 x 11 x 11 portion

    // Dewit
    setPathVolumeKernel<<<dimGrid, dimBlock>>>(d_pv, d_dv, pv->height,
      pv->width, pv->depth);
  }

  // Copy the path volume back into host memory
  hipMemcpy(pv->contents, d_pv, fvDataLen * sizeof(float),
    hipMemcpyDeviceToHost);

  // Clear memory
  hipFree(d_dv);
  hipFree(d_pv);
}

void setBigPathVolumeParallel(struct FloatVolume *pv, struct FloatVolume *dv,
  unsigned subvolume_height) {
  struct FloatVolume spv, sdv;
  unsigned subvolume_size, numIterations, i, lastIterHeight;
  float *y0buf;

  // Initialize the empty sub-pathvolume
  setEmptyFloatVolume(&spv, subvolume_height, dv->width, dv->depth);
  // Initialize the empty sub-diffvolume
  setEmptyFloatVolume(&sdv, subvolume_height, dv->width, dv->depth);
  // Initialize the empty final path volume
  setEmptyFloatVolume(pv, dv->height, dv->width, dv->depth);

  // Buffer holding the last y = 0 data
  y0buf = (float *) malloc(sizeof(float) * spv.width * spv.depth);

  // Creating the path volume will be done increments of sub-pathvolumes
  subvolume_size = subvolume_height * dv->width * dv->depth;
  // Find out how many sub-pathvolumes we will need to calculate
  // Every subvolume will calculate subvolume_size - 1 portion of the
  //   total subvolume, since the y = 0 of the subvolume is already in the
  //   total subvolume
  printf("subvolume_height: %u\n", subvolume_height);
  printf("pv->height - 1: %u\n", pv->height - 1);
  printf("spv.height - 1: %u\n", spv.height - 1);
  numIterations = (pv->height - 1) / (spv.height - 1);
  if((pv->height - 1) % (spv.height - 1)) numIterations++;

  // Set the very first cell in the final path volume
  *(pv->contents + 0) = *(dv->contents + 0);

  // Complete x = 0, y = 0
  setX0Y0(pv, dv);
  // Complete z = 0, y = 0
  setZ0Y0(pv, dv);
  // Complete y = 0
  setY0(pv, dv);

  for(i = 0; i < numIterations; i++) {
    // Resize the subvolumes if it is the last iteration
    if(i == numIterations - 1) {
      // Get the height of the volume of the last iteration
      lastIterHeight = ((pv->height - 1) % (spv.height - 1)) + 1;

      // Change dimensions of subvolumes
      spv.height = lastIterHeight;
      sdv.height = lastIterHeight;

      // Reallocate each of the subvolumes
      free(spv.contents);
      free(sdv.contents);
      spv.contents = (float *) malloc(sizeof(float) * spv.height * spv.width *
        spv.depth);
      sdv.contents = (float *) malloc(sizeof(float) * sdv.height * sdv.width *
        sdv.depth);
    }

    // Set the contents of the subvolumes

    /* // The path subvolume
    if(i == 0) {
      // Copy y = 0 to sub-pathvolume
      memcpy(spv.contents, pv->contents, sizeof(float) * spv.width * spv.depth);
    } else {
      // Copy y = spv.height - 1 from sub-pathvolume to y = 0 from
      // sub-pathvolume
      memcpy(spv.contents, y0buf, sizeof(float) * spv.width * spv.depth);
    } */

    // The sub diffvolume
    //memcpy(sdv.contents, dv->contents + subvolume_size, sizeof(float) *
    //  subvolume_size);
    memcpy(sdv.contents, dv->contents + (sdv.height - 1) * sdv.width *
      sdv.depth * i, sizeof(float) * sdv.height * sdv.width * sdv.depth);

    printf("--- sub-diffvolume %u ---\n", i);
    printFloatVolume(&sdv);

    /* // Complete path subvolume
    setSmallPathVolumeParallel(&spv, &sdv);

    // Copy the contents of the path subvolume to the total volume
    memcpy(pv->contents + pv->width * pv->depth + i *
      (spv.height - 1) * spv.width * spv.depth,
      spv.contents + spv.width * spv.depth,
      sizeof(float) * (spv.height - 1) * spv.width * spv.depth);

    // Copy the contents at y = max y to y = 0 within the path subvolume
    memcpy(y0buf, spv.contents + (spv.height - 1) * spv.width * spv.height,
      sizeof(float) * spv.width * spv.height); */
  }

  /* printf("--- State check ---\n");
  printf("subvolume_size: %u\n", subvolume_size);
  printf("numIterations: %u\n", numIterations);
  printf("-- sub-pathvolume --\n");
  printFloatVolume(&spv);
  printf("-- sub-diffvolume --\n");
  printFloatVolume(&sdv); */

  // Deallocation
  free(y0buf);
  free(spv.contents);
  free(sdv.contents);
}

int main() {
  struct Picture picture1, picture2;
  struct FloatVolume dvs, dvp;
  struct FloatVolume pvs, pvp;
  unsigned i, j, res;

  srand(time(NULL));

  for(i = 4; i < 25; i++) {
  for(j = 4; j < 25; j++) {

  printf("(%u, %u)\n", i , j);

  // --- PICTURE CREATION SECTION ---------------------------------------------

  /*setRandomPicture(&picture1, 6, 6);
  setRandomPicture(&picture2, 6, 6);*/
  setRandomPicture(&picture1, i, j);
  setRandomPicture(&picture2, i, j);

  printf("--- picture1 ---\n");
  /* printPicture(&picture1);
  printf("\n"); */

  printf("--- picture2 ---\n");
  /* printPicture(&picture2);
  printf("\n"); */

  // --- DIFF VOLUME SECTION --------------------------------------------------

  setDiffVolumeSerial(&dvs, &picture1, &picture2);

  printf("--- diff volume serial ---\n");
  /* printFloatVolume(&dvs);
  printf("\n"); */

  // setDiffVolumeParallel(&dvp, &picture1, &picture2);
  setBigDiffVolumeParallel(&dvp, &picture1, &picture2, 2);

  printf("--- diff volume parallel ---\n");
  /* printFloatVolume(&dvp);
  printf("\n"); */

  printf("--- diff volume comparison ---\n");
  res = compareFloatVolumes(&dvs, &dvp);
  printf("%d\n", res);
  if(res != 0)
    exit(1);
  //printf("%d\n", compareFloatVolumes(&dvs, &dvp));
  printf("\n");

  // --- PATH VOLUME SECTION --------------------------------------------------

  setPathVolumeSerial(&pvs, &dvs);

  printf("--- path volume serial ---\n");
  /* printFloatVolume(&pvs);
  printf("\n"); */

  //setBigPathVolumeParallel(&pvp, &dvp, 3);
  // Print test volume
  /* printf("-- test volume --\n");
  printFloatVolume(&pvp); */
  // Deallocate test volume
  // free(pvp.contents);

  setPathVolumeParallel(&pvp, &dvp);

  printf("--- path volume parallel ---\n");
  /* printFloatVolume(&pvp);
  printf("\n"); */

  printf("--- path volume comparison ---\n");
  res = compareFloatVolumes(&pvs, &pvp);
  printf("%d\n", res);
  if(res != 0)
    exit(1);
  //printf("%d\n", compareFloatVolumes(&pvs, &pvp));
  printf("\n");

  // --- DEALLOCATION ---------------------------------------------------------

  free(picture1.colors);
  free(picture2.colors);
  free(dvs.contents);
  free(dvp.contents);
  free(pvs.contents);
  free(pvp.contents);

  }
  }
}
